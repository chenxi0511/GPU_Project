#include "hip/hip_runtime.h"
/*
 * Naive Parallel version with shared mempry.
 * The performance gain is about 2.5 times compared to the naive parallelization.
 */
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "bitmap.h"

//define blocksize
#define BLOCKSIZE 1024
#define PIX_KEY_WIDTH 32
#define PIX_KEY_HEIGHT 32
#define IMAGESIZE 512*512

__device__ int col_calculator(int, int);
__device__ int row_calculator(int, int);
__device__ int de_key_generator(int, int, int);

void swap (int *a, int *b)
{
    int temp = *a;
    *a = *b;
    *b = temp;
}

void randomize ( int arr[], int n )
{
	//randomly switch add[i] with another element with random index
    srand(time(NULL));
    for (int i = n -1; i > 0; i--)
    {
        int j = rand() % (i+1);
        swap(&arr[i], &arr[j]);
    }
}


void substitution_key_generator(int *sub_key)
{
    time_t t;
    srand((unsigned) time(&t));
    for(int i = 0; i < BLOCKSIZE; i++){
        sub_key[i] = rand() % 256;
    }
}

void encryption_permutation_key_generator(int *per_key, int size)
{
    for(int i = 0; i < size; i++){
        per_key[i] = i+1;
    }
}

//Step 1: Pixel substitution kernel
__global__ void substitution(unsigned char *inputImage, int *sub_key,int imagewidth)
{
    int block_x = threadIdx.x % PIX_KEY_WIDTH;
    int block_y = threadIdx.x / PIX_KEY_WIDTH;
    
    int block_idx_x = blockIdx.x % (imagewidth / PIX_KEY_WIDTH);  
    int block_idx_y = blockIdx.x / (imagewidth / PIX_KEY_WIDTH);   
    
    int idx_pixel = (block_idx_y * PIX_KEY_HEIGHT + block_y) * imagewidth + (block_idx_x * PIX_KEY_WIDTH + block_x);

    __shared__ unsigned char s_data[BLOCKSIZE * 3];
    __shared__ int s_key[BLOCKSIZE];

    for(int i = 0; i < 3; i++)
    {
    	s_data[threadIdx.x * 3 + i] = inputImage[idx_pixel * 3 + i];
    }

    s_key[threadIdx.x] = sub_key[threadIdx.x];
    __syncthreads();

    for(int i=0;i<3;i++)
    {
    	s_data[threadIdx.x * 3 + i] = s_data[threadIdx.x * 3 + i] ^ s_key[threadIdx.x];
		inputImage[idx_pixel * 3 + i] = s_data[threadIdx.x * 3+i];
    }
}

__device__ int col_calculator(int key, int col)
{
    int N = col;
    return key % N;
}

__device__ int row_calculator(int key, int row)
{   
    int N = row;
    return key / N + 1;
}

__device__ int de_key_generator(int col, int row, int width)
{   
    return row * width + col + 1; 
}

//Step 2: Pixel permutation within one block kernel
__global__ void pixel_permutation(unsigned char *inputImage, unsigned char *outputImage,
		int *en_key, int *de_key,int imagewidth)
{
    int block_x = threadIdx.x % PIX_KEY_WIDTH;
    int block_y = threadIdx.x / PIX_KEY_WIDTH;
    
    int block_idx_x = blockIdx.x % (imagewidth/PIX_KEY_WIDTH);   
    int block_idx_y = blockIdx.x / (imagewidth/PIX_KEY_WIDTH);   
    
    int idx_pixel = (block_idx_y * PIX_KEY_HEIGHT + block_y) * imagewidth + (block_idx_x * PIX_KEY_WIDTH + block_x);

    __shared__ unsigned char s_data[BLOCKSIZE*3];
    __shared__ unsigned char s_data_out[BLOCKSIZE*3];
    __shared__ int s_data_en_key[BLOCKSIZE];
    __shared__ int s_data_de_key[BLOCKSIZE];

    for(int i = 0; i < 3; i++)
    {
    	s_data[threadIdx.x * 3 + i] = inputImage[idx_pixel * 3 + i];
    }
    s_data_en_key[threadIdx.x] = en_key[threadIdx.x];
    __syncthreads();
  
    int key = s_data_en_key[threadIdx.x];
    int new_col = col_calculator(key, PIX_KEY_WIDTH);
    int new_row = row_calculator(key, PIX_KEY_WIDTH);
    if(new_col == 0)
    {
        new_col = PIX_KEY_WIDTH;
        new_row -= 1;
    }
    s_data_de_key[(new_row - 1) * PIX_KEY_WIDTH + (new_col - 1)]
                  = de_key_generator(block_x, block_y, PIX_KEY_WIDTH);

    for(int i = 0; i < 3; i++)
    {
    	s_data_out[((new_row - 1) * PIX_KEY_WIDTH + (new_col - 1)) * 3 + i]
    	           = s_data[threadIdx.x * 3 + i];
    }
    __syncthreads();

    for(int i = 0; i < 3; i++)
    {
    	outputImage[idx_pixel * 3 + i] = s_data_out[threadIdx.x * 3 + i];
    }
    de_key[threadIdx.x] = s_data_de_key[threadIdx.x];
}

//Step 3: Block permutation kernel
__global__ void block_permutation(unsigned char *InputImage, unsigned char *outputImage,
		int *en_key, int *de_key,int imagewidth)
{
/***
// it seems that the shared memory does not speed up in the block permutation
    int block_x = threadIdx.x % PIX_KEY_WIDTH;
    int block_y = threadIdx.x / PIX_KEY_WIDTH;

    int block_idx_x = blockIdx.x % (imagewidth/PIX_KEY_WIDTH);
    int block_idx_y = blockIdx.x / (imagewidth/PIX_KEY_WIDTH);

    int idx_pixel=(block_idx_y * PIX_KEY_HEIGHT + block_y) * imagewidth + (block_idx_x * PIX_KEY_WIDTH + block_x);

    __shared__ unsigned char s_data[BLOCKSIZE*3];
    __shared__ int s_data_en_key[IMAGESIZE/BLOCKSIZE];
    __shared__ int s_data_de_key[IMAGESIZE/BLOCKSIZE];

    for(int i = 0; i < 3; i++)
    {
    	s_data[threadIdx.x * 3 + i] = InputImage[idx_pixel * 3 + i];
    }
    s_data_en_key[blockIdx.x] = en_key[blockIdx.x];
    __syncthreads();

    int key = s_data_en_key[blockIdx.x];
    int new_col_block=  col_calculator(key, imagewidth/PIX_KEY_WIDTH);
    int new_row_block = row_calculator(key, imagewidth/PIX_KEY_WIDTH);
    if(new_col_block == 0)
    {
        new_col_block = imagewidth/PIX_KEY_WIDTH;
        new_row_block -= 1;
    }
    s_data_de_key[(new_row_block - 1) * imagewidth/PIX_KEY_WIDTH + (new_col_block - 1)]
    	= de_key_generator(block_idx_x, block_idx_y, imagewidth/PIX_KEY_WIDTH);

    int idx_pixel_toperm=((new_row_block-1) * PIX_KEY_HEIGHT) * imagewidth + (new_col_block-1) * PIX_KEY_WIDTH + block_y * imagewidth + block_x;
    for(int i = 0; i < 3; i++)
    {
    	outputImage[idx_pixel_toperm * 3 + i] = s_data[threadIdx.x * 3 + i];
    }

    de_key[(new_row_block - 1) * imagewidth/PIX_KEY_WIDTH + (new_col_block - 1)]
    	= s_data_de_key[(new_row_block - 1) * imagewidth/PIX_KEY_WIDTH + (new_col_block - 1)];
    __syncthreads();
***/

    int block_x = threadIdx.x % PIX_KEY_WIDTH;	// col position in block
    int block_y = threadIdx.x / PIX_KEY_WIDTH;	// row position in block

    int block_idx_x = blockIdx.x % (imagewidth / PIX_KEY_WIDTH);   //block position in the image
    int block_idx_y = blockIdx.x / (imagewidth / PIX_KEY_WIDTH);

    int idx_pixel = (block_idx_y * PIX_KEY_HEIGHT + block_y) * imagewidth + (block_idx_x * PIX_KEY_WIDTH + block_x);

    int key = en_key[block_idx_y * (imagewidth/PIX_KEY_WIDTH) + block_idx_x];
    int new_col_block = col_calculator(key, imagewidth / PIX_KEY_WIDTH);
    int new_row_block = row_calculator(key, imagewidth / PIX_KEY_WIDTH);
    if(new_col_block == 0)
    {
        new_col_block = imagewidth / PIX_KEY_WIDTH;
        new_row_block -= 1;
    }

    de_key[(new_row_block - 1) * (imagewidth / PIX_KEY_WIDTH) + (new_col_block - 1)]
           = de_key_generator(block_idx_x, block_idx_y, imagewidth / PIX_KEY_WIDTH);
    for(int i = 0; i < 3; i++)
    {
        outputImage[(((new_row_block-1) * PIX_KEY_HEIGHT) * imagewidth + (new_col_block-1) * PIX_KEY_WIDTH +(block_y * imagewidth+ block_x)) * 3 + i]
                    = InputImage[idx_pixel * 3 + i];
    }
}

int main(int argc, char *argv[])
{
    INFOHEADER bitmapInfoHeader;
    FILEHEADER bitmapFileHeader;
    unsigned char *input_image;
    unsigned char *pixel_permutation_image;
    unsigned char *block_permutation_iamge;

    INFOHEADER bitmapInfoHeader1;
    FILEHEADER bitmapFileHeader1;
    unsigned char *output_image;
    unsigned char *out_block_permutation_iamge;
    unsigned char *out_pixel_permutation_image;

    unsigned char *d_input_image;
    unsigned char *d_pixel_permutation_image;
    unsigned char *d_block_permutation_iamge;
    unsigned char *d_output_image;
    unsigned char *d_out_block_permutation_iamge;
    unsigned char *d_out_pixel_permutation_image;

    
    //Loading input image and allocate enough memory for processing
    input_image = LoadImage(argv[1],&bitmapInfoHeader, &bitmapFileHeader);
    pixel_permutation_image = (unsigned char *) malloc(bitmapInfoHeader.imagesize * sizeof(char));
    memset(pixel_permutation_image, 0, bitmapInfoHeader.imagesize);
    block_permutation_iamge = (unsigned char *) malloc(bitmapInfoHeader.imagesize * sizeof(char));
    memset(block_permutation_iamge, 0, bitmapInfoHeader.imagesize);
    
    printf("Image size: %d\n", bitmapInfoHeader.imagesize);
    printf("Image widthh: %d\n", bitmapInfoHeader.width);
    printf("Image height: %d\n", bitmapInfoHeader.height);
    int imagewidth = bitmapInfoHeader.width;
    
    int substitution_key[ BLOCKSIZE];
    substitution_key_generator(substitution_key);
    int *d_substitution_key;

    int pix_perm_key[BLOCKSIZE];
    int *d_pix_per_key;
    int *d_pix_per_key_de;
    encryption_permutation_key_generator(pix_perm_key, BLOCKSIZE);
    randomize(pix_perm_key, BLOCKSIZE);

    int block_perm_key[IMAGESIZE/BLOCKSIZE];
    int *d_block_per_key;
    int *d_block_per_key_de;
    encryption_permutation_key_generator(block_perm_key,IMAGESIZE/BLOCKSIZE);
    randomize(block_perm_key, IMAGESIZE/BLOCKSIZE);
   
    //CUDA malloc and CUDA memcpy
    hipMalloc((void**)&d_input_image, IMAGESIZE*3);
    hipMalloc((void**)&d_pixel_permutation_image, IMAGESIZE*3);
    hipMalloc((void**)&d_block_permutation_iamge, IMAGESIZE*3);

    hipMalloc((void**)&d_substitution_key, BLOCKSIZE * sizeof(int));
    hipMalloc((void**)&d_pix_per_key, BLOCKSIZE * sizeof(int));
    hipMalloc((void**)&d_pix_per_key_de, BLOCKSIZE * sizeof(int));
    hipMalloc((void**)&d_block_per_key, IMAGESIZE/BLOCKSIZE* sizeof(int));
    hipMalloc((void**)&d_block_per_key_de, IMAGESIZE/BLOCKSIZE * sizeof(int));

    hipMemcpy(d_input_image, input_image, IMAGESIZE*3, hipMemcpyHostToDevice);
    hipMemcpy(d_pixel_permutation_image, pixel_permutation_image, IMAGESIZE*3, hipMemcpyHostToDevice);
    hipMemcpy(d_block_permutation_iamge, block_permutation_iamge, IMAGESIZE*3, hipMemcpyHostToDevice);

    hipMemcpy(d_substitution_key, substitution_key, BLOCKSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pix_per_key, pix_perm_key, BLOCKSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_pix_per_key_de, 0, BLOCKSIZE * sizeof(int));
    hipMemcpy(d_block_per_key, block_perm_key, IMAGESIZE/BLOCKSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_block_per_key_de, 0,IMAGESIZE/BLOCKSIZE * sizeof(int));

    //define the dimension of CUDA grid and block
    dim3 gridDim(IMAGESIZE/BLOCKSIZE , 1, 1);
    dim3 blockDim(BLOCKSIZE, 1, 1);

    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t end;
    hipEventCreate(&end);
    float en_substitution_time, en_pixel_permutation_time, en_block_permutation_time;
    float de_substitution_time, de_pixel_permutation_time, de_block_permutation_time;

    //Image Encryption using three CUDA kernels
    hipEventRecord(start, 0);
    substitution<<<gridDim, blockDim>>>(d_input_image, d_substitution_key, imagewidth);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&en_substitution_time, start, end);
    printf("Encryption substitution Time: %fms\n", en_substitution_time);

    hipEventRecord(start, 0);
    pixel_permutation<<<gridDim, blockDim>>>(d_input_image, d_pixel_permutation_image, d_pix_per_key, d_pix_per_key_de, imagewidth);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&en_pixel_permutation_time, start, end);
    printf("Encryption pixel permutation Time: %fms\n", en_pixel_permutation_time);

    hipEventRecord(start, 0);
    block_permutation<<<gridDim, blockDim>>>(d_pixel_permutation_image, d_block_permutation_iamge, d_block_per_key, d_block_per_key_de, imagewidth);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&en_block_permutation_time, start, end);
    printf("Encryption block permutation Time: %fms\n", en_block_permutation_time);
    printf("Encryption time: %fms\n", en_substitution_time + en_pixel_permutation_time + en_block_permutation_time);

    hipMemcpy(block_permutation_iamge, d_block_permutation_iamge, IMAGESIZE*3, hipMemcpyDeviceToHost);
    SaveImage(argv[2], block_permutation_iamge, &bitmapFileHeader, &bitmapInfoHeader);

    
    output_image = LoadImage(argv[2],&bitmapInfoHeader1, &bitmapFileHeader1);
    out_block_permutation_iamge = (unsigned char *) malloc(bitmapInfoHeader.imagesize * sizeof(char));
    memset(out_block_permutation_iamge, 0, IMAGESIZE);
    out_pixel_permutation_image = (unsigned char *) malloc(bitmapInfoHeader.imagesize * sizeof(char));
    memset(out_pixel_permutation_image, 0, IMAGESIZE);

    hipMalloc((void**)&d_output_image, IMAGESIZE*3);
    hipMalloc((void**)&d_out_block_permutation_iamge, IMAGESIZE*3);
    hipMalloc((void**)&d_out_pixel_permutation_image, IMAGESIZE*3);

    hipMemcpy(d_output_image, output_image, IMAGESIZE*3, hipMemcpyHostToDevice);
    hipMemcpy(d_out_block_permutation_iamge, out_block_permutation_iamge, IMAGESIZE*3, hipMemcpyHostToDevice);
    hipMemcpy(d_out_pixel_permutation_image, out_pixel_permutation_image, IMAGESIZE*3, hipMemcpyHostToDevice);

    //Image Decryption using three CUDA kernels
    hipEventRecord(start, 0);
    block_permutation<<<gridDim, blockDim>>>(d_output_image, d_out_block_permutation_iamge, d_block_per_key_de, d_block_per_key,imagewidth);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&de_block_permutation_time, start, end);
    printf("Decryption block permutation Time: %fms\n", de_block_permutation_time);

    hipEventRecord(start, 0);
    pixel_permutation<<<gridDim, blockDim>>>(d_out_block_permutation_iamge, d_out_pixel_permutation_image, d_pix_per_key_de, d_pix_per_key,imagewidth);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&de_pixel_permutation_time, start, end);
    printf("Decryption pixel permutation Time: %fms\n", de_pixel_permutation_time);

    hipEventRecord(start, 0);
    substitution<<<gridDim, blockDim>>>(d_out_pixel_permutation_image, d_substitution_key,imagewidth);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&de_substitution_time, start, end);
    printf("Decryption substitution Time: %fms\n", de_substitution_time);
    printf("Decryption time: %fms\n", de_substitution_time + de_pixel_permutation_time + de_block_permutation_time);

    hipMemcpy(out_pixel_permutation_image,  d_out_pixel_permutation_image, IMAGESIZE*3, hipMemcpyDeviceToHost);
    SaveImage(argv[3], out_pixel_permutation_image, &bitmapFileHeader1, &bitmapInfoHeader1);

    //Memory free
    hipFree(d_input_image);
    hipFree(d_pixel_permutation_image);
    hipFree(d_block_permutation_iamge);
    hipFree(d_output_image);
    hipFree(d_out_block_permutation_iamge);
    hipFree(d_out_pixel_permutation_image);

    hipFree(d_substitution_key);
    hipFree(d_pix_per_key);
    hipFree(d_pix_per_key_de);
    hipFree(d_block_per_key);
    hipFree(d_block_per_key_de);

    return 0;
}
